#include "hip/hip_runtime.h"
// TODO: handle hipMallocManaged failures
// TODO: query max block size
#include "neural_net.h"

#include "matrix.h"
#include "matrix.cpp"

#include "matrix_test.cpp"

#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
#include <assert.h>

__device__
float CudaGetMatrixElement(matrix* Matrix, uint32_t Row, uint32_t Column)
{
	assert(Row < Matrix->NumRows);
	assert(Column < Matrix->NumColumns);
	float* Element = Matrix->Data + Row * Matrix->NumColumns + Column;
	return *Element;
}

__device__
float CudaGetMatrixElement(matrix* Matrix, uint32_t ElementIndex)
{
	// NOTE: made available if the Row, Column asserts in the standard 
	// CONT: GetMatrixElement isn't needed. Mostly used for when you don't care
	// CONT: if you have a row or column matrix
	assert(ElementIndex < (Matrix->NumRows * Matrix->NumColumns));
	float* Element = Matrix->Data + ElementIndex;
	return *Element;
}

__device__
void CudaSetMatrixElement(
	matrix* Matrix, uint32_t Row, uint32_t Column, float Value
)
{
	assert(Row < Matrix->NumRows);
	assert(Column < Matrix->NumColumns);
	float* Element = Matrix->Data + Row * Matrix->NumColumns + Column;
	*Element = Value;
}

void CudaInitMatrix(matrix* Matrix, uint32_t NumRows, uint32_t NumColumns)
{
	*Matrix = {};
	Matrix->NumRows = NumRows;
	Matrix->NumColumns = NumColumns;
	hipMallocManaged(&Matrix->Data, GetMatrixDataSize(Matrix));
	memset(Matrix->Data, 0, GetMatrixDataSize(Matrix));
}

void CudaAllocMatrix(matrix** Result, uint32_t NumRows, uint32_t NumColumns)
{
	hipMallocManaged(Result, sizeof(matrix));
	matrix* Matrix = *Result;
	CudaInitMatrix(Matrix, NumRows, NumColumns);
}

void CudaAllocMultResultMatrix(matrix** Result, matrix* M1, matrix* M2)
{
	// NOTE: allocates a matrix that would result from the matrix multiplication
	// CONT: of M1 and M2
	CudaAllocMatrix(Result, M1->NumRows, M2->NumColumns);
}

void CudaAllocM1TransposeMultResultMatrix(
	matrix** Result, matrix* M1, matrix* M2
)
{
	// NOTE: allocates a matrix that would result from the matrix multiplication
	// CONT: of M1 tranposed and M2
	CudaAllocMatrix(Result, M1->NumColumns, M2->NumColumns);
}

void CudaAllocM2TransposeMultResultMatrix(
	matrix** Result, matrix* M1, matrix* M2
)
{
	// NOTE: allocates a matrix that would result from the matrix multiplication
	// CONT: of M1 tranposed and M2
	CudaAllocMatrix(Result, M1->NumRows, M2->NumRows);
}

void CudaAllocM1M2TransposeMultResultMatrix(
	matrix** Result, matrix* M1, matrix* M2
)
{
	// NOTE: allocates a matrix that would result from the matrix multiplication
	// CONT: of M1 tranposed and M2
	CudaAllocMatrix(Result, M1->NumColumns, M2->NumRows);
}

void CudaAllocMatrixMeanResult(matrix** Result, matrix* M1)
{
	CudaAllocMatrix(Result, 1, M1->NumColumns);
}

// TODO: get cuda memory free in here
// void FreeMatrixData(matrix Matrix)
// {
// 	free(Matrix.Data);
// }

// void FreeMatrix(matrix* Matrix)
// {
// 	FreeMatrixData(*Matrix);
// 	free(Matrix);
// }

inline int GetNumBlocks(int Range, int BlockSize)
{
	// TODO: query for max block size?
	return (Range + BlockSize - 1) / BlockSize;
}

__global__
void CudaMatrixMultCore(matrix* M1, matrix* M2, matrix* Result)
{	
	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	int Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	int Stride = blockDim.x * gridDim.x;

	for(uint32_t Row = Start; Row < M1->NumRows; Row += Stride)
	{
		for(uint32_t Column = 0; Column < M2->NumColumns; Column++)
		{
			float DotProduct = 0.0f;
			for(uint32_t DPIndex = 0; DPIndex < M1->NumColumns; DPIndex++)
			{
				DotProduct += (
					CudaGetMatrixElement(M1, Row, DPIndex) * 
					CudaGetMatrixElement(M2, DPIndex, Column)
				);
			}
			CudaSetMatrixElement(Result, Row, Column, DotProduct);
		}
	}
}

void CudaMatrixMult(matrix* M1, matrix* M2, matrix* Result)
{
	assert(M1->NumColumns == M2->NumRows);
	// NOTE: not sure if this should be a variable or queried or tracked with 
	// CONT: a data structure
	int BlockSize = 256;

	// NOTE: NumBlocks is always at least one, and grows as the data to 
	// NOTE: process grows
	int NumBlocks = GetNumBlocks(M1->NumRows, BlockSize);
	CudaMatrixMultCore<<<NumBlocks, BlockSize>>>(M1, M2, Result);
	hipDeviceSynchronize();
}

__global__
void CudaAddVectorToRowsCore(matrix* M1, matrix* Vector, matrix* Result)
{
	/*NOTE:
	Because the vector is one-dimensional, it doesn't matter whether you pass 
	Col into the row or the column 
	a nice consequence of this is that it doesn't matter whether you pass in a 
	row vector or a column vector. It will project nicely as long as the non-one
	dimension is equal to the number of columns of M1
	*/

	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	int Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	int Stride = blockDim.x * gridDim.x;

	for(uint32_t Row = Start; Row < M1->NumRows; Row += Stride)
	{
		for(uint32_t Col = 0; Col < M1->NumColumns; Col++)
		{
			CudaSetMatrixElement(
				Result,
				Row,
				Col,
				(
					CudaGetMatrixElement(M1, Row, Col) + 
					CudaGetMatrixElement(Vector, Col)
				)
			);
		}
	}
}

void CudaAddVectorToRows(matrix* M1, matrix* Vector, matrix* Result)
{
	// NOTE: this function is equivalent to adding two matrices, M1 and M2,
	// CONT: where M2 has the same values in each row (Vector) 
	// NOTE: there's no reason to allocate a huge matrix just for this, so this 
	// CONT: method is used instead
	assert(
		(M1->NumColumns == Vector->NumColumns) ||
		(M1->NumColumns == Vector->NumRows)
	);

	// NOTE: not sure if this should be a variable or queried or tracked with 
	// CONT: a data structure
	int BlockSize = 256;
	int NumBlocks = GetNumBlocks(M1->NumRows, BlockSize);
	CudaAddVectorToRowsCore<<<NumBlocks, BlockSize>>>(M1, Vector, Result);
	hipDeviceSynchronize();
}

__global__
void CudaMatrixAddCore(matrix* M1, matrix* M2, matrix* Result)
{
	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	int Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	int Stride = blockDim.x * gridDim.x;

	for(uint32_t Row = Start; Row < M1->NumRows; Row += Stride)
	{
		for(uint32_t Col = 0; Col < M1->NumColumns; Col++)
		{
			CudaSetMatrixElement(
				Result,
				Row,
				Col,
				CudaGetMatrixElement(M1, Row, Col) + 
				CudaGetMatrixElement(M2, Row, Col)
			);
		}
	}
}

void CudaMatrixAdd(matrix* M1, matrix* M2, matrix* Result)
{
	assert(M1->NumRows == M2->NumRows);
	assert(M1->NumColumns == M2->NumColumns);
	
	// NOTE: not sure if this should be a variable or queried or tracked with 
	// CONT: a data structure
	int BlockSize = 256;
	int NumBlocks = GetNumBlocks(M1->NumRows, BlockSize);
	CudaMatrixAddCore<<<NumBlocks, BlockSize>>>(M1, M2, Result);
	hipDeviceSynchronize();
}

__global__
void CudaMatrixMultM1TransposeCore(matrix* M1, matrix* M2, matrix* Result)
{
	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	int Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	int Stride = blockDim.x * gridDim.x;

	for(uint32_t Row = Start; Row < M1->NumColumns; Row += Stride)
	{
		for(uint32_t Column = 0; Column < M2->NumColumns; Column++)
		{
			float DotProduct = 0.0f;
			for(uint32_t DPIndex = 0; DPIndex < M1->NumRows; DPIndex++)
			{
				DotProduct += (
					CudaGetMatrixElement(M1, DPIndex, Row) * 
					CudaGetMatrixElement(M2, DPIndex, Column)
				);
			}
			CudaSetMatrixElement(Result, Row, Column, DotProduct);
		}
	}
}

void CudaMatrixMultM1Transpose(matrix* M1, matrix* M2, matrix* Result)
{
	// NOTE: For transpose multiplication without allocating and initializing
	// CONT: a new matrix
	// NOTE: the number of rows in M1 should equal the number of rows in M2

	assert(M1->NumRows == M2->NumRows);

	int BlockSize = 256;
	int NumBlocks = GetNumBlocks(M1->NumColumns, BlockSize);
	CudaMatrixMultM1TransposeCore<<<NumBlocks, BlockSize>>>(M1, M2, Result);
	hipDeviceSynchronize();
}

__global__
void CudaMatrixMultM2TransposeCore(matrix* M1, matrix* M2, matrix* Result)
{
	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	int Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	int Stride = blockDim.x * gridDim.x;

	for(uint32_t Row = Start; Row < M1->NumRows; Row += Stride)
	{
		for(uint32_t Column = 0; Column < M2->NumRows; Column++)
		{
			float DotProduct = 0.0f;
			for(uint32_t DPIndex = 0; DPIndex < M1->NumColumns; DPIndex++)
			{
				DotProduct += (
					CudaGetMatrixElement(M1, Row, DPIndex) * 
					CudaGetMatrixElement(M2, Column, DPIndex)
				);
			}
			CudaSetMatrixElement(Result, Row, Column, DotProduct);
		}
	}
}

void CudaMatrixMultM2Transpose(matrix* M1, matrix* M2, matrix* Result)
{
	// NOTE: For transpose multiplication without allocating and initializing
	// CONT: a new matrix
	// NOTE: the number of columns in M1 should equal the number of columns in M2

	assert(M1->NumColumns == M2->NumColumns);

	int BlockSize = 256;
	int NumBlocks = GetNumBlocks(M1->NumRows, BlockSize);
	CudaMatrixMultM2TransposeCore<<<NumBlocks, BlockSize>>>(M1, M2, Result);
	hipDeviceSynchronize();
}

__global__
void CudaMatrixMultM1M2TransposeCore(matrix* M1, matrix* M2, matrix* Result)
{
	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	int Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	int Stride = blockDim.x * gridDim.x;

	for(uint32_t Row = Start; Row < M1->NumColumns; Row += Stride)
	{
		for(uint32_t Column = 0; Column < M2->NumRows; Column++)
		{
			float DotProduct = 0.0f;
			for(uint32_t DPIndex = 0; DPIndex < M1->NumRows; DPIndex++)
			{
				DotProduct += (
					CudaGetMatrixElement(M1, DPIndex, Row) * 
					CudaGetMatrixElement(M2, Column, DPIndex)
				);
			}
			CudaSetMatrixElement(Result, Row, Column, DotProduct);
		}
	}
}

void CudaMatrixMultM1M2Transpose(matrix* M1, matrix* M2, matrix* Result)
{
	// NOTE: For transpose multiplication without allocating and initializing
	// CONT: a new matrix
	assert(M1->NumRows == M2->NumColumns);

	int BlockSize = 256;
	int NumBlocks = GetNumBlocks(M1->NumColumns, BlockSize);
	CudaMatrixMultM1M2TransposeCore<<<NumBlocks, BlockSize>>>(M1, M2, Result);
	hipDeviceSynchronize();
}

__global__
void CudaMatrixScalarMultCore(float Scalar, matrix* M1, matrix* Result)
{
	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	int Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	int Stride = blockDim.x * gridDim.x;

	for(uint32_t Row = Start; Row < M1->NumRows; Row += Stride)
	{
		for(uint32_t Column = 0; Column < M1->NumColumns; Column++)
		{
			float NewValue = Scalar * CudaGetMatrixElement(M1, Row, Column);
			CudaSetMatrixElement(Result, Row, Column, NewValue);
		}
	}
}

void CudaMatrixScalarMult(float Scalar, matrix* M1, matrix* Result)
{
	int BlockSize = 256;
	int NumBlocks = GetNumBlocks(M1->NumRows, BlockSize);
	CudaMatrixScalarMultCore<<<NumBlocks, BlockSize>>>(Scalar, M1, Result);
	hipDeviceSynchronize();
}

__global__
void CudaMatrixSubtractCore(matrix* M1, matrix* M2, matrix* Result)
{
	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	int Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	int Stride = blockDim.x * gridDim.x;

	for(uint32_t Row = Start; Row < M1->NumRows; Row += Stride)
	{
		for(uint32_t Col = 0; Col < M1->NumColumns; Col++)
		{
			CudaSetMatrixElement(
				Result,
				Row,
				Col,
				CudaGetMatrixElement(M1, Row, Col) - 
				CudaGetMatrixElement(M2, Row, Col)
			);
		}
	}
}

void CudaMatrixSubtract(matrix* M1, matrix* M2, matrix* Result)
{
	assert(M1->NumRows == M2->NumRows);
	assert(M1->NumColumns == M2->NumColumns);

	int BlockSize = 256;
	int NumBlocks = GetNumBlocks(M1->NumRows, BlockSize);

	CudaMatrixSubtractCore<<<BlockSize, NumBlocks>>>(M1, M2, Result);
	hipDeviceSynchronize();
}

__device__
void CudaMatrixScalarMultCoreColStride(
	float Scalar, matrix* M1, matrix* Result, int Start, int Stride
)
{
	// NOTE: the number of columns in M1 should equal the number of rows in M2
	// NOTE: mostly a helper function for the mean function
	for(uint32_t Row = 0; Row < M1->NumRows; Row++)
	{
		for(uint32_t Column = Start; Column < M1->NumColumns; Column += Stride)
		{
			float NewValue = Scalar * CudaGetMatrixElement(M1, Row, Column);
			CudaSetMatrixElement(Result, Row, Column, NewValue);
		}
	}
}

__global__
void CudaMatrixMeanCore(matrix* M1, matrix* Result)
{
	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	int Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	int Stride = blockDim.x * gridDim.x;

	CudaMatrixScalarMultCoreColStride(0.0f, Result, Result, Start, Stride);
	for(uint32_t Row = 0; Row < M1->NumRows; Row++)
	{
		for(uint32_t Col = Start; Col < M1->NumColumns; Col += Stride)
		{
			float NewValue = (
				CudaGetMatrixElement(Result, 0, Col) + 
				CudaGetMatrixElement(M1, Row, Col)
			);
			CudaSetMatrixElement(Result, 0, Col, NewValue);
		}
	}
	CudaMatrixScalarMultCoreColStride(
		1.0f / M1->NumRows, Result, Result, Start, Stride
	);
}

void CudaMatrixMean(matrix* M1, matrix* Result)
{
	/*NOTE:
	This function finds the sum of all the row vectors of matrix M1 and divides
	that sum by the number of rows. 

	M1 Dimensions: N x M
	Result Dimensions: 1 x M
	*/
	int BlockSize = 256;
	int NumBlocks = GetNumBlocks(M1->NumColumns, BlockSize);
	CudaMatrixMeanCore<<<NumBlocks, BlockSize>>>(M1, Result);
	hipDeviceSynchronize();
}

void CudaAllocDenseLayer(
	dense_layer** Result, uint32_t InputDim, uint32_t OutputDim
)
{
	hipMallocManaged(Result, sizeof(dense_layer));
	dense_layer* DenseLayer = *Result;
	*DenseLayer = {};
	CudaInitMatrix(&DenseLayer->Weights, InputDim, OutputDim);
	CudaInitMatrix(&DenseLayer->Bias, 1, OutputDim);
}

void CudaDenseForward(matrix* Inputs, dense_layer* DenseLayer, matrix* Results)
{
	CudaMatrixMult(Inputs, &DenseLayer->Weights, Results);
	CudaAddVectorToRows(Results, &DenseLayer->Bias, Results);	
}

void CudaAllocDenseLayerTrain(
	dense_layer_train_data** Result,
	dense_layer* DenseLayer,
	float LearningRate,
	uint32_t BatchSize
)
{
	hipMallocManaged(Result, sizeof(dense_layer_train_data));
	dense_layer_train_data* TrainData = *Result;
	*TrainData = {};
	TrainData->LearningRate = LearningRate; 
	CudaInitMatrix(
		&TrainData->WeightsDelta,
		DenseLayer->Weights.NumRows,
		DenseLayer->Weights.NumColumns
	);
	CudaInitMatrix(
		&TrainData->BiasDelta,
		DenseLayer->Bias.NumRows,
		DenseLayer->Bias.NumColumns
	);
	CudaInitMatrix(
		&TrainData->LayerGradient, BatchSize, DenseLayer->Weights.NumRows
	);
}

void CudaDenseBack(
	matrix* Inputs,
	matrix* NextLayerGradient,
	dense_layer* DenseLayer,
	dense_layer_train_data* TrainData
)
{
	CudaMatrixMultM2Transpose(
		NextLayerGradient, &DenseLayer->Weights, &TrainData->LayerGradient
	);

	CudaMatrixMultM1Transpose(
		Inputs, NextLayerGradient, &TrainData->WeightsDelta
	);
	CudaMatrixScalarMult(
		TrainData->LearningRate,
		&TrainData->WeightsDelta,
		&TrainData->WeightsDelta
	);
	CudaMatrixAdd(
		&DenseLayer->Weights,
		&TrainData->WeightsDelta,
		&DenseLayer->Weights
	);
	
	CudaMatrixMean(NextLayerGradient, &TrainData->BiasDelta);
	CudaMatrixScalarMult(
		TrainData->LearningRate,
		&TrainData->BiasDelta,
		&TrainData->BiasDelta
	);
	CudaMatrixAdd(
		&DenseLayer->Bias,
		&TrainData->BiasDelta,
		&DenseLayer->Bias
	);
}

void CudaAllocReluTrain(
	relu_train_data** Result, uint32_t BatchSize, uint32_t InputDim
)
{
	hipMallocManaged(Result, sizeof(relu_train_data));
	relu_train_data* TrainData = *Result;
	*TrainData = {};
	CudaInitMatrix(&TrainData->LayerGradient, BatchSize, InputDim);
}

// TODO: implement free
// void FreeReluTrain(relu_train_data* TrainData)
// {
// 	FreeMatrixData(TrainData->LayerGradient);
// 	free(TrainData);
// }

__global__
void CudaReluForwardCore(matrix* M1, matrix* Result)
{
	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	int Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	int Stride = blockDim.x * gridDim.x;

	for(uint32_t Row = Start; Row < M1->NumRows; Row += Stride)
	{
		for(uint32_t Col = 0; Col < M1->NumColumns; Col++)
		{
			float NewValue;
			float OldValue = CudaGetMatrixElement(M1, Row, Col);
			if(OldValue < 0)
			{
				NewValue = 0;
			}
			else
			{
				NewValue = OldValue;
			}
			CudaSetMatrixElement(Result, Row, Col, NewValue);
		}
	}
}


void CudaReluForward(matrix* Inputs, matrix* Outputs)
{
	assert(Inputs->NumRows == Outputs->NumRows);
	assert(Inputs->NumColumns == Outputs->NumColumns);

	int BlockSize = 256;
	int NumBlocks = GetNumBlocks(Inputs->NumRows, BlockSize);
	CudaReluForwardCore<<<NumBlocks, BlockSize>>>(Inputs, Outputs);
	hipDeviceSynchronize();
}

__global__
void CudaReluBackCore(
	matrix* Inputs, matrix* NextLayerGradient, matrix* LayerGradient
)
{
	uint32_t Start = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t Stride = gridDim.x * blockDim.x;
	for(uint32_t Row = Start; Row < Inputs->NumRows; Row += Stride)
	{
		for(uint32_t Col = 0; Col < Inputs->NumColumns; Col++)
		{
			float LayerGradientElement;
			float InputValue = CudaGetMatrixElement(Inputs, Row, Col);
			if(InputValue <= 0)
			{
				LayerGradientElement = 0;
			}
			else
			{
				LayerGradientElement = CudaGetMatrixElement(
					NextLayerGradient, Row, Col
				);
			}
			CudaSetMatrixElement(LayerGradient, Row, Col, LayerGradientElement);
		}
	}
}

void CudaReluBack(
	matrix* Inputs, matrix* NextLayerGradient, relu_train_data* TrainData
)
{
	int BlockSize = 256;
	int NumBlocks = GetNumBlocks(Inputs->NumRows, BlockSize);
	CudaReluBackCore<<<NumBlocks, BlockSize>>>(
		Inputs, NextLayerGradient, &TrainData->LayerGradient
	);
	hipDeviceSynchronize();
}

struct mean_squared_layer
{
	int MaxThreads;
	float* SquaredErrorResults;
};

void CudaAllocMeanSquared(mean_squared_layer** Result, int MaxThreads)
{
	hipMallocManaged(Result, sizeof(mean_squared_layer));
	mean_squared_layer* Layer = *Result;
	*Layer = {};
	Layer->MaxThreads = MaxThreads;
	hipMallocManaged(&Layer->SquaredErrorResults, MaxThreads * sizeof(float));
}

__global__
void CudaMeanSquaredForwardCore(
	float* SquaredErrorResults, matrix* Predictions, matrix* Labels
)
{
	int Start = blockIdx.x * blockDim.x + threadIdx.x;
	int Stride = gridDim.x * blockDim.x;

	float Result = 0.0f;
	for(uint32_t Row = Start; Row < Predictions->NumRows; Row += Stride)
	{
		for(uint32_t Col = 0; Col < Predictions->NumColumns; Col++)
		{
			float Difference = (
				CudaGetMatrixElement(Predictions, Row, Col) - 
				CudaGetMatrixElement(Labels, Row, Col)
			);
			Result += Difference * Difference;
		}
	}
	float* SquaredError = SquaredErrorResults + Start;
	*SquaredError = Result;
}

float CudaMeanSquaredForward(
	mean_squared_layer* Layer, matrix* Predictions, matrix* Labels
)
{
	int BlockSize = 256;
	int NumBlocks = GetNumBlocks(Predictions->NumRows, BlockSize);
	assert((NumBlocks * BlockSize) < Layer->MaxThreads);
	memset(Layer->SquaredErrorResults, 0, Layer->MaxThreads * sizeof(float));
	CudaMeanSquaredForwardCore<<<NumBlocks, BlockSize>>>(
		Layer->SquaredErrorResults, Predictions, Labels
	);
	hipDeviceSynchronize();
		
	int NumThreadsRan; 
	if(Layer->MaxThreads < Predictions->NumRows)
	{
		NumThreadsRan = Layer->MaxThreads;
	}
	else
	{
		NumThreadsRan = Predictions->NumRows;
	}

	float Sum = 0;
	for(
		uint32_t ThreadIndex = 0;
		ThreadIndex < NumThreadsRan;
		ThreadIndex++
	)
	{
		float* SquaredError = Layer->SquaredErrorResults + ThreadIndex;
		Sum += *SquaredError;
	}

	// NOTE: this definition of MSE with a two in the denominator helps cancel 
	// CONT: out a two in the back derivation 
	float Mean = Sum / (2 * Predictions->NumRows);
	return Mean;
}

void CudaAllocMseTrainData(
	mse_train_data** Result, uint32_t BatchSize, uint32_t PredictionDim
)
{
	hipMallocManaged(Result, sizeof(mse_train_data));
	mse_train_data* TrainData = *Result;
	*TrainData = {};
	CudaInitMatrix(&TrainData->LayerGradient, BatchSize, PredictionDim);
}

// TODO: implement me
// void FreeMseTrainData(mse_train_data* TrainData)
// {
// 	FreeMatrixData(TrainData->LayerGradient);
// 	free(TrainData);
// }

void CudaMeanSquaredBack(
	matrix* Predictions, matrix* Labels, mse_train_data* TrainData
)
{
	CudaMatrixSubtract(
		Labels, Predictions, &TrainData->LayerGradient
	);
	CudaMatrixScalarMult(
		1.0f / Predictions->NumColumns,
		&TrainData->LayerGradient,
		&TrainData->LayerGradient
	);
}


#define SAVE_RESULTS 0
matrix* TestMatrixResult(
	matrix* M1,
	char* FilePathBuffer,
	size_t FilePathBufferSize,
	char* TestDataDirectory,
	const char* TestName,
	char* EndianString
)
{
	// NOTE: if this function is changed in both test programs 3 more times, 
	// CONT: it's time to refactor it

	snprintf(
		FilePathBuffer,
		FilePathBufferSize,
		"%s/%s_%s.data",
		TestDataDirectory,
		TestName,
		EndianString
	);
#if SAVE_RESULTS
	SaveMatrix(M1, FilePathBuffer);
#endif

	matrix* CompareTo;
	CudaAllocMatrix(&CompareTo, M1->NumRows, M1->NumColumns);
	bool LoadResult = LoadMatrix(CompareTo, FilePathBuffer);
	if(!LoadResult)
	{
		printf("Could not read %s\n", FilePathBuffer);
	}
	else if(!MatricesAreEquivalent(M1, CompareTo))
	{
		printf("%s failed\n", TestName);
		printf("Expected\n");
		PrintMatrix(CompareTo);
		printf("Got\n");
		PrintMatrix(M1);
	}

	return CompareTo;
}

void TestFloatResult(
	float Result,
	char* FilePathBuffer,
	size_t FilePathBufferSize,
	char* TestDataDirectory,
	const char* TestName,
	char* EndianString
)
{
	snprintf(
		FilePathBuffer,
		FilePathBufferSize,
		"%s/%s_%s.data",
		TestDataDirectory,
		TestName,
		EndianString
	);
	FILE* File;
#if SAVE_RESULTS
	fopen_s(&File, FilePathBuffer, "w");
	fwrite(&Result, 1, sizeof(float), File);
	fclose(File);
#endif 
	float Expected;
	fopen_s(&File, FilePathBuffer, "r");
	fread(&Expected, 1, sizeof(float), File);
	fclose(File);

	if(Expected != Result)
	{
		printf("Failure in %s\n", TestName);
	}
}

int main(int argc, char* argv[])
{
	// TODO: move test code out to other file

	char TestDataDirectory[260];
	if(argc == 1)
	{
		printf("Assuming test data directory path is ../test_data\n");
		strcpy_s(TestDataDirectory, sizeof(TestDataDirectory), "../test_data");
	}
	else if(argc > 1)
	{
		strcpy_s(TestDataDirectory, sizeof(TestDataDirectory), argv[1]);
		printf("TestDataDirectory is %s\n", TestDataDirectory);
	}
	else
	{
		return -1;
	}

	bool BigEndian = IsBigEndian();
	char EndianString[260];
	if(BigEndian)
	{
		strcpy_s(EndianString, sizeof(EndianString), "BigEndian");
	}
	else
	{
		strcpy_s(EndianString, sizeof(EndianString), "LittleEndian");
	}
	char FilePathBuffer[260];

	// SECTION START: Matrix tests
	{
		matrix* M1;
		uint32_t NumRows = 3;
		uint32_t NumColumns = 3;
		CudaAllocMatrix(&M1, NumRows, NumColumns);
		FillMatrixConsecutive(M1);		

		matrix* M2;
		NumRows = 3;
		NumColumns = 3;
		CudaAllocMatrix(&M2, NumRows, NumColumns);
		FillMatrixConsecutive(M2);

		matrix* MultResult;
		CudaAllocMultResultMatrix(&MultResult, M1, M2);
		CudaMatrixMult(M1, M2, MultResult);
		// NOTE: TestMatrixResult returns a matrix pointer that can be freed
		TestMatrixResult(
			MultResult,
			FilePathBuffer,
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaMultResult",
			EndianString
		);

		MatrixClear(MultResult);

		matrix* M3;
		NumRows = 3;
		NumColumns = 2;
		CudaAllocMatrix(&M3, NumRows, NumColumns);
		FillMatrixConsecutive(M3);

		matrix* M4;
		NumRows = 2;
		NumColumns = 3;
		CudaAllocMatrix(&M4, NumRows, NumColumns);
		FillMatrixConsecutive(M4);

		matrix* MultResult2;
		CudaAllocMultResultMatrix(&MultResult2, M3, M4);
		CudaMatrixMult(M3, M4, MultResult2);
		TestMatrixResult(
			MultResult2,
			FilePathBuffer, 
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaNonSquareMult",
			EndianString
		);

		matrix* AddResult;
		CudaAllocMatrix(&AddResult, M1->NumRows, M1->NumColumns);
		CudaMatrixAdd(M1, M2, AddResult);
		TestMatrixResult(
			AddResult,
			FilePathBuffer, 
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaMatrixAdd",
			EndianString
		);

		matrix* AddVectorResult;
		CudaAllocMatrix(&AddVectorResult, M1->NumRows, M1->NumColumns);
		matrix* Vector;
		CudaAllocMatrix(&Vector, 1, M1->NumColumns);
		FillMatrixConsecutive(Vector);
		CudaAddVectorToRows(M1, Vector, AddVectorResult);
		TestMatrixResult(
			AddVectorResult,
			FilePathBuffer, 
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaAddVectorToRows",
			EndianString
		);

		matrix* M5;
		NumRows = 2;
		NumColumns = 3;
		CudaAllocMatrix(&M5, NumRows, NumColumns);
		FillMatrixConsecutive(M5);

		matrix* M6;
		NumRows = 2;
		NumColumns = 3;
		CudaAllocMatrix(&M6, NumRows, NumColumns);
		FillMatrixConsecutive(M6);

		matrix* M5TMultResult;
		CudaAllocM1TransposeMultResultMatrix(&M5TMultResult, M5, M6);
		CudaMatrixMultM1Transpose(M5, M6, M5TMultResult);
		TestMatrixResult(
			M5TMultResult,
			FilePathBuffer, 
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaMatrixMultM1Transpose",
			EndianString
		);

		MatrixClear(M5TMultResult);
		SetMatrixElement(M6, 0, 1, 7);
		SetMatrixElement(M6, 1, 2, 13);
		
		CudaMatrixMultM1Transpose(M5, M6, M5TMultResult);
		TestMatrixResult(
			M5TMultResult,
			FilePathBuffer, 
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaNonSymmetricMatrixMultM1Transpose",
			EndianString
		);

		matrix* M6TMultResult;
		CudaAllocM2TransposeMultResultMatrix(&M6TMultResult, M5, M6);

		CudaMatrixMultM2Transpose(M5, M6, M6TMultResult);
		TestMatrixResult(
			M6TMultResult,
			FilePathBuffer,
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaMatrixMultM2Transpose",
			EndianString
		);

		matrix* M7;
		NumRows = 2;
		NumColumns = 3;
		CudaAllocMatrix(&M7, NumRows, NumColumns);
		FillMatrixConsecutive(M7);

		matrix* M8;
		NumRows = 3;
		NumColumns = 2;
		CudaAllocMatrix(&M8, NumRows, NumColumns);
		FillMatrixConsecutive(M8);

		matrix* M7TM8TMultResult;
		CudaAllocM1M2TransposeMultResultMatrix(&M7TM8TMultResult, M7, M8);
		MatrixClear(M7TM8TMultResult);
		CudaMatrixMultM1M2Transpose(M7, M8, M7TM8TMultResult);
		TestMatrixResult(
			M7TM8TMultResult,
			FilePathBuffer, 
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaMatrixMultM1M2Transpose",
			EndianString
		);

		matrix* M9;
		NumRows = 3;
		NumColumns = 4;
		CudaAllocMatrix(&M9, NumRows, NumColumns);
		FillMatrixConsecutive(M9);
		
		CudaMatrixScalarMult(0.5f, M9, M9);
		TestMatrixResult(
			M9,
			FilePathBuffer, 
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaMatrixScalarMult",
			EndianString
		);

		matrix* M10;
		NumRows = 4;
		NumColumns = 4;
		CudaAllocMatrix(&M10, NumRows, NumColumns);
		FillMatrixConsecutive(M10);

		matrix* M10Mean;
		CudaAllocMatrixMeanResult(&M10Mean, M10);
		MatrixClear(M10Mean);
		CudaMatrixMean(M10, M10Mean);
		TestMatrixResult(
			M10Mean,
			FilePathBuffer, 
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaMatrixRowMean",
			EndianString
		);

		matrix* M11;
		NumRows = 3;
		NumColumns = 4;
		CudaAllocMatrix(&M11, NumRows, NumColumns);
		FillMatrixConsecutive(M11);

		matrix* M12;
		CudaAllocMatrix(&M12, NumRows, NumColumns);
		FillMatrixConsecutive(M12);
		SetMatrixElement(M12, 0, 0, -2.0f);
		matrix* SubResult;
		CudaAllocMatrix(&SubResult, NumRows, NumColumns);
		CudaMatrixSubtract(M11, M12, SubResult);

		TestMatrixResult(
			SubResult,
			FilePathBuffer,
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaMatrixSub",
			EndianString
		);
		// NOTE: if memory starts getting hefty, free memory here
	}
	// SECTION STOP: Matrix tests

	// SECTION START: Dense layer tests
	{
		uint32_t BatchSize = 8;
		uint32_t InputDim = 4;
		uint32_t OutputDim = 3;
		matrix* Inputs;
		CudaAllocMatrix(&Inputs, BatchSize, InputDim);
		FillMatrixConsecutive(Inputs);

		matrix* Outputs;
		CudaAllocMatrix(&Outputs, BatchSize, OutputDim);
		MatrixClear(Outputs);

		dense_layer* DenseLayer;
		CudaAllocDenseLayer(&DenseLayer, InputDim, OutputDim);
		FillMatrixConsecutive(&DenseLayer->Weights);
		FillMatrixConsecutive(&DenseLayer->Bias);
		CudaDenseForward(Inputs, DenseLayer, Outputs);
		TestMatrixResult(
			Outputs,
			FilePathBuffer, 
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaForwardDense",
			EndianString
		);

		matrix* NextLayerGradient;
		CudaAllocMatrix(&NextLayerGradient, BatchSize, OutputDim);
		FillMatrixConsecutive(NextLayerGradient);

		dense_layer_train_data* TrainData;
		CudaAllocDenseLayerTrain(&TrainData, DenseLayer, 1.0f, BatchSize);
		CudaDenseBack(
			Inputs, NextLayerGradient, DenseLayer, TrainData
		);
		TestMatrixResult(
			&DenseLayer->Weights,
			FilePathBuffer, 
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaDenseWeightsAfterUpdate",
			EndianString
		);
		TestMatrixResult(
			&DenseLayer->Bias,
			FilePathBuffer, 
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaDenseBiasAfterUpdate",
			EndianString
		);
		TestMatrixResult(
			&TrainData->LayerGradient,
			FilePathBuffer, 
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaDenseLayerGradient",
			EndianString
		);
	}
	// SECTION STOP: Dense layer tests

	// SECTION START: RELU tests
	{
		uint32_t BatchSize = 8;
		uint32_t InputDim = 4;

		matrix* Inputs;
		CudaAllocMatrix(&Inputs, BatchSize, InputDim);
		FillMatrixConsecutive(Inputs);

		matrix* Outputs;
		CudaAllocMatrix(&Outputs, BatchSize, InputDim);
		CudaReluForward(Inputs, Outputs);
		TestMatrixResult(
			Outputs,
			FilePathBuffer, 
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaReluForwardPositive",
			EndianString
		);

		matrix* NextLayerGradient;
		CudaAllocMatrix(&NextLayerGradient, BatchSize, InputDim);
		FillMatrixConsecutive(NextLayerGradient);

		relu_train_data* TrainData;
		CudaAllocReluTrain(&TrainData, BatchSize, InputDim);
		CudaReluBack(Inputs, NextLayerGradient, TrainData);
		TestMatrixResult(
			&TrainData->LayerGradient,
			FilePathBuffer, 
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaReluLayerGradientPositive",
			EndianString
		);

		CudaMatrixScalarMult(-1.0f, Inputs, Inputs);
		CudaReluForward(Inputs, Outputs);
		TestMatrixResult(
			Outputs,
			FilePathBuffer, 
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaReluForwardNegative",
			EndianString
		);

		CudaReluBack(Inputs, NextLayerGradient, TrainData);
		TestMatrixResult(
			&TrainData->LayerGradient,
			FilePathBuffer, 
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaReluLayerGradientNegative",
			EndianString
		);
	}
	// SECTION STOP: RELU Tests

	// SECTION START: MSE Test
	{
		uint32_t BatchSize = 8;
		uint32_t NumClasses = 4;

		matrix* Predictions = NULL;
		CudaAllocMatrix(&Predictions, BatchSize, NumClasses);
		FillOneHotMatrix(Predictions);
		
		matrix* Labels = NULL; 
		CudaAllocMatrix(&Labels, BatchSize, NumClasses);
		FillOneHotMatrix(Labels);

		mean_squared_layer* MseLayer = NULL;
		CudaAllocMeanSquared(&MseLayer, 1 << 14);

		float Loss = CudaMeanSquaredForward(MseLayer, Predictions, Labels);
		TestFloatResult(
			Loss,
			FilePathBuffer,
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaMSELoss",
			EndianString
		);

		mse_train_data* TrainData = NULL;
		CudaAllocMseTrainData(&TrainData, BatchSize, NumClasses);
		CudaMeanSquaredBack(Predictions, Labels, TrainData);
		TestMatrixResult(
			&TrainData->LayerGradient,
			FilePathBuffer, 
			sizeof(FilePathBuffer),
			TestDataDirectory,
			"CudaMSEBackOK",
			EndianString
		);
	}
	// SECTION STOP: MSE Test

	// TODO: maybe add another MSE test with non-zero resulting loss and 
	// CONT: layer gradient
	return 0;
}