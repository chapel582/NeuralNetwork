#include "hip/hip_runtime.h"
// TODO: handle hipMallocManaged failures
// TODO: query max block size
#include "arg_max.h"
#include "int_shuffler.h"
#include "neural_net.h"
#include "matrix.h"
#include "neural_net_cpu.h"

#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

void CudaInitMatrix(matrix* Matrix, uint32_t NumRows, uint32_t NumColumns)
{
	*Matrix = {};
	Matrix->NumRows = NumRows;
	Matrix->NumColumns = NumColumns;
	hipMallocManaged(&Matrix->Data, GetMatrixDataSize(Matrix));
	memset(Matrix->Data, 0, GetMatrixDataSize(Matrix));
}

void CudaFreeMatrixData(matrix* Matrix)
{
	hipFree(Matrix->Data);
}

void CudaAllocMatrix(matrix** Result, uint32_t NumRows, uint32_t NumColumns)
{
	hipMallocManaged(Result, sizeof(matrix));
	matrix* Matrix = *Result;
	CudaInitMatrix(Matrix, NumRows, NumColumns);
}

void CudaFreeMatrix(matrix* Matrix)
{
	CudaFreeMatrixData(Matrix);
	hipFree(Matrix);
}

void CudaAllocMultResultMatrix(matrix** Result, matrix* M1, matrix* M2)
{
	// NOTE: allocates a matrix that would result from the matrix multiplication
	// CONT: of M1 and M2
	CudaAllocMatrix(Result, M1->NumRows, M2->NumColumns);
}

void CudaAllocM1TransposeMultResultMatrix(
	matrix** Result, matrix* M1, matrix* M2
)
{
	// NOTE: allocates a matrix that would result from the matrix multiplication
	// CONT: of M1 tranposed and M2
	CudaAllocMatrix(Result, M1->NumColumns, M2->NumColumns);
}

void CudaAllocM2TransposeMultResultMatrix(
	matrix** Result, matrix* M1, matrix* M2
)
{
	// NOTE: allocates a matrix that would result from the matrix multiplication
	// CONT: of M1 tranposed and M2
	CudaAllocMatrix(Result, M1->NumRows, M2->NumRows);
}

void CudaAllocM1M2TransposeMultResultMatrix(
	matrix** Result, matrix* M1, matrix* M2
)
{
	// NOTE: allocates a matrix that would result from the matrix multiplication
	// CONT: of M1 tranposed and M2
	CudaAllocMatrix(Result, M1->NumColumns, M2->NumRows);
}

void CudaAllocMatrixMeanResult(matrix** Result, matrix* M1)
{
	CudaAllocMatrix(Result, 1, M1->NumColumns);
}

#define MAX_GPUS 1
int GlobalMaxBlockSizeArray[MAX_GPUS];
int GlobalMaxGridDimArray[MAX_GPUS];

void CudaInitDeviceProperties(uint32_t Device)
{
	assert(Device < MAX_GPUS);
	hipDeviceGetAttribute(
		&GlobalMaxBlockSizeArray[Device], hipDeviceAttributeMaxBlockDimX, Device
	);

	// TODO: the value returned seems to be either wrong or pointless
	hipDeviceGetAttribute(
		&GlobalMaxGridDimArray[Device], hipDeviceAttributeMaxGridDimX, Device
	);
	GlobalMaxGridDimArray[Device] = 64;
}

uint32_t GetBlockSize(uint32_t Device)
{
	return GlobalMaxBlockSizeArray[Device];
}

uint32_t GetMaxNumBlocks(uint32_t Device)
{
	return GlobalMaxGridDimArray[Device];
}

uint32_t GetNumBlocks(uint32_t Range, uint32_t BlockSize, uint32_t Device)
{
	// NOTE: for getting max blocks for operations that are parallelizable 
	// CONT: without no sync

	// NOTE: NumBlocks is always at least one, and grows as the data to 
	// CONT: process grows
	
	uint32_t NumBlocks = (Range + BlockSize - 1) / BlockSize;
	int MaxBlocks = GetMaxNumBlocks(Device);
	if(MaxBlocks < NumBlocks)
	{
		MaxBlocks = NumBlocks;
	}
	return NumBlocks;
}

__global__
void CudaMatrixMultThread(matrix* M1, matrix* M2, matrix* Result)
{	
	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	uint32_t Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	uint32_t Stride = blockDim.x * gridDim.x;

	MatrixMultCore(M1, M2, Result, Start, Stride);
}

void CudaMatrixMult(matrix* M1, matrix* M2, matrix* Result)
{
	assert(M1->NumColumns == M2->NumRows);
	uint32_t Device = 0;
	int BlockSize = GetBlockSize(Device);
	int NumBlocks = GetNumBlocks(
		GetMatrixArrayCount(Result), BlockSize, Device
	);
	CudaMatrixMultThread<<<NumBlocks, BlockSize>>>(M1, M2, Result);
	hipDeviceSynchronize();
}

__global__
void CudaAddVectorToRowsThread(matrix* M1, matrix* Vector, matrix* Result)
{
	/*NOTE:
	Because the vector is one-dimensional, it doesn't matter whether you pass 
	Col into the row or the column 
	a nice consequence of this is that it doesn't matter whether you pass in a 
	row vector or a column vector. It will project nicely as long as the non-one
	dimension is equal to the number of columns of M1
	*/

	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	uint32_t Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	uint32_t Stride = blockDim.x * gridDim.x;

	AddVectorToRowsCore(M1, Vector, Result, Start, Stride);
}

void CudaAddVectorToRows(matrix* M1, matrix* Vector, matrix* Result)
{
	// NOTE: this function is equivalent to adding two matrices, M1 and M2,
	// CONT: where M2 has the same values in each row (Vector) 
	// NOTE: there's no reason to allocate a huge matrix just for this, so this 
	// CONT: method is used instead
	assert(
		(M1->NumColumns == Vector->NumColumns) ||
		(M1->NumColumns == Vector->NumRows)
	);

	// NOTE: not sure if this should be a variable or queried or tracked with 
	// CONT: a data structure
	int Device = 0;
	uint32_t BlockSize = GetBlockSize(Device);
	uint32_t NumBlocks = GetNumBlocks(
		GetMatrixArrayCount(M1), BlockSize, Device
	);
	CudaAddVectorToRowsThread<<<NumBlocks, BlockSize>>>(M1, Vector, Result);
	hipDeviceSynchronize();
}

__global__
void CudaMatrixAddThread(matrix* M1, matrix* M2, matrix* Result)
{
	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	uint32_t Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	uint32_t Stride = blockDim.x * gridDim.x;

	MatrixAddCore(M1, M2, Result, Start, Stride);
}

void CudaMatrixAdd(matrix* M1, matrix* M2, matrix* Result)
{
	assert(M1->NumRows == M2->NumRows);
	assert(M1->NumColumns == M2->NumColumns);
	
	// NOTE: not sure if this should be a variable or queried or tracked with 
	// CONT: a data structure
	int Device = 0;
	uint32_t BlockSize = GetBlockSize(Device);
	uint32_t NumBlocks = GetNumBlocks(
		GetMatrixArrayCount(M1), BlockSize, Device
	);
	CudaMatrixAddThread<<<NumBlocks, BlockSize>>>(M1, M2, Result);
	hipDeviceSynchronize();
}

__global__
void CudaMatrixMultM1TransposeThread(matrix* M1, matrix* M2, matrix* Result)
{
	uint32_t Start = blockIdx.x * blockDim.x + threadIdx.x;  
	uint32_t Stride = blockDim.x * gridDim.x;
	
	MatrixMultM1TransposeCore(M1, M2, Result, Start, Stride);
}

void CudaMatrixMultM1Transpose(matrix* M1, matrix* M2, matrix* Result)
{
	// NOTE: For transpose multiplication without allocating and initializing
	// CONT: a new matrix
	// NOTE: the number of rows in M1 should equal the number of rows in M2

	assert(M1->NumRows == M2->NumRows);

	int Device = 0;
	uint32_t BlockSize = GetBlockSize(Device);
	uint32_t NumBlocks = GetNumBlocks(
		GetMatrixArrayCount(Result), BlockSize, Device
	);
	CudaMatrixMultM1TransposeThread<<<NumBlocks, BlockSize>>>(M1, M2, Result);
	hipDeviceSynchronize();
}

__global__
void CudaMatrixMultM2TransposeThread(matrix* M1, matrix* M2, matrix* Result)
{
	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	uint32_t Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	uint32_t Stride = blockDim.x * gridDim.x;

	MatrixMultM2TransposeCore(M1, M2, Result, Start, Stride);
}

void CudaMatrixMultM2Transpose(matrix* M1, matrix* M2, matrix* Result)
{
	// NOTE: For transpose multiplication without allocating and initializing
	// CONT: a new matrix
	// NOTE: the number of columns in M1 should equal the number of columns in M2

	assert(M1->NumColumns == M2->NumColumns);

	int Device = 0;
	uint32_t BlockSize = GetBlockSize(Device);
	uint32_t NumBlocks = GetNumBlocks(
		GetMatrixArrayCount(Result), BlockSize, Device
	);
	CudaMatrixMultM2TransposeThread<<<NumBlocks, BlockSize>>>(M1, M2, Result);
	hipDeviceSynchronize();
}

__global__
void CudaMatrixMultM1M2TransposeThread(matrix* M1, matrix* M2, matrix* Result)
{
	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	uint32_t Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	uint32_t Stride = blockDim.x * gridDim.x;

	MatrixMultM1M2TransposeCore(M1, M2, Result, Start, Stride);
}

void CudaMatrixMultM1M2Transpose(matrix* M1, matrix* M2, matrix* Result)
{
	// NOTE: For transpose multiplication without allocating and initializing
	// CONT: a new matrix
	assert(M1->NumRows == M2->NumColumns);

	int Device = 0;
	uint32_t BlockSize = GetBlockSize(Device);
	uint32_t NumBlocks = GetNumBlocks(
		GetMatrixArrayCount(Result), BlockSize, Device
	);
	CudaMatrixMultM1M2TransposeThread<<<NumBlocks, BlockSize>>>(M1, M2, Result);
	hipDeviceSynchronize();
}

__global__
void CudaMatrixScalarMultThread(float Scalar, matrix* M1, matrix* Result)
{
	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	uint32_t Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	uint32_t Stride = blockDim.x * gridDim.x;

	MatrixScalarMultCore(Scalar, M1, Result, Start, Stride);
}

void CudaMatrixScalarMult(float Scalar, matrix* M1, matrix* Result)
{
	int Device = 0;
	uint32_t BlockSize = GetBlockSize(Device);
	uint32_t NumBlocks = GetNumBlocks(
		GetMatrixArrayCount(M1), BlockSize, Device
	);
	CudaMatrixScalarMultThread<<<NumBlocks, BlockSize>>>(Scalar, M1, Result);
	hipDeviceSynchronize();
}

__global__
void CudaMatrixSubtractThread(matrix* M1, matrix* M2, matrix* Result)
{
	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	uint32_t Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	uint32_t Stride = blockDim.x * gridDim.x;

	MatrixSubtractCore(M1, M2, Result, Start, Stride);
}

void CudaMatrixSubtract(matrix* M1, matrix* M2, matrix* Result)
{
	assert(M1->NumRows == M2->NumRows);
	assert(M1->NumColumns == M2->NumColumns);

	int Device = 0;
	uint32_t BlockSize = GetBlockSize(Device);
	uint32_t NumBlocks = GetNumBlocks(
		GetMatrixArrayCount(M1), BlockSize, BlockSize
	);

	CudaMatrixSubtractThread<<<NumBlocks, BlockSize>>>(M1, M2, Result);
	hipDeviceSynchronize();
}

__global__
void CudaMatrixMeanThread(matrix* M1, matrix* Result)
{
	// NOTE: this basically indexes by the thread index, but b/c the thread 
	// CONT: index is reset on every block, 
	uint32_t Start = blockIdx.x * blockDim.x + threadIdx.x;  
	// NOTE: this basically calculates the # of threads
	uint32_t Stride = blockDim.x * gridDim.x;

	MatrixMeanCore(M1, Result, Start, Stride);
}

void CudaMatrixMean(matrix* M1, matrix* Result)
{
	/*NOTE:
	This function finds the sum of all the row vectors of matrix M1 and divides
	that sum by the number of rows. 

	M1 Dimensions: N x M
	Result Dimensions: 1 x M
	*/
	int Device = 0;
	uint32_t BlockSize = GetBlockSize(Device);
	uint32_t NumBlocks = GetNumBlocks(M1->NumColumns, BlockSize, Device);
	CudaMatrixMeanThread<<<NumBlocks, BlockSize>>>(M1, Result);
	hipDeviceSynchronize();
}

void CudaAllocDenseLayer(
	dense_layer** Result, uint32_t InputDim, uint32_t OutputDim
)
{
	hipMallocManaged(Result, sizeof(dense_layer));
	dense_layer* DenseLayer = *Result;
	*DenseLayer = {};
	CudaInitMatrix(&DenseLayer->Weights, InputDim, OutputDim);
	CudaInitMatrix(&DenseLayer->Bias, 1, OutputDim);
}

void CudaFreeDenseLayer(dense_layer* DenseLayer)
{
	CudaFreeMatrixData(&DenseLayer->Weights);
	CudaFreeMatrixData(&DenseLayer->Bias);
	hipFree(DenseLayer);
}

__device__
void CudaDenseForwardCore(
	matrix* Inputs,
	dense_layer* DenseLayer,
	matrix* Results,
	uint32_t Start,
	uint32_t Stride
)
{
	MatrixMultCore(Inputs, &DenseLayer->Weights, Results, Start, Stride);
	__syncthreads();
	AddVectorToRowsCore(
		Results, &DenseLayer->Bias, Results, Start, Stride
	);
}

__global__
void CudaDenseForwardThread(
	matrix* Inputs, dense_layer* DenseLayer, matrix* Results
)
{
	uint32_t Start = blockIdx.x * blockDim.x + threadIdx.x;  
	uint32_t Stride = blockDim.x * gridDim.x;

	CudaDenseForwardCore(Inputs, DenseLayer, Results, Start, Stride);
}

void CudaDenseForward(matrix* Inputs, dense_layer* DenseLayer, matrix* Results)
{
	int Device = 0;
	uint32_t BlockSize = GetBlockSize(Device);
	uint32_t NumBlocks = GetNumBlocks(
		GetMatrixArrayCount(Results), BlockSize, Device
	);
	CudaDenseForwardThread<<<NumBlocks, BlockSize>>>(
		Inputs, DenseLayer, Results
	);
	hipDeviceSynchronize();
}

void CudaAllocDenseLayerTrain(
	dense_layer_train_data** Result,
	dense_layer* DenseLayer,
	float LearningRate,
	uint32_t BatchSize
)
{
	hipMallocManaged(Result, sizeof(dense_layer_train_data));
	dense_layer_train_data* TrainData = *Result;
	*TrainData = {};
	TrainData->LearningRate = LearningRate; 
	CudaInitMatrix(
		&TrainData->WeightsDelta,
		DenseLayer->Weights.NumRows,
		DenseLayer->Weights.NumColumns
	);
	CudaInitMatrix(
		&TrainData->BiasDelta,
		DenseLayer->Bias.NumRows,
		DenseLayer->Bias.NumColumns
	);
	CudaInitMatrix(
		&TrainData->LayerGradient, BatchSize, DenseLayer->Weights.NumRows
	);
}

void CudaFreeDenseLayerTrain(dense_layer_train_data* TrainData)
{
	CudaFreeMatrixData(&TrainData->WeightsDelta);
	CudaFreeMatrixData(&TrainData->BiasDelta);
	CudaFreeMatrixData(&TrainData->LayerGradient);
	hipFree(TrainData);
}

__device__
void CudaDenseBackCore(
	matrix* Inputs,
	matrix* NextLayerGradient,
	dense_layer* DenseLayer,
	dense_layer_train_data* TrainData,
	uint32_t Start,
	uint32_t Stride
)
{
	// NOTE: all of these operations don't have any dependencies on other 
	// CONT: thread's outcomes, so we don't need calls to __syncthreads 

	matrix* Weights = &DenseLayer->Weights;

	// NOTE: Calculate this layer's gradient
	MatrixMultM2TransposeCore(
		NextLayerGradient,
		Weights,
		&TrainData->LayerGradient,
		Start,
		Stride
	);
	__syncthreads();

	// NOTE: Calculate the delta for the weights
	matrix* WeightsDelta = &TrainData->WeightsDelta;
	MatrixMultM1TransposeCore(
		Inputs, NextLayerGradient, WeightsDelta, Start, Stride
	);
	__syncthreads();
	MatrixScalarMultCore(
		TrainData->LearningRate, WeightsDelta, WeightsDelta, Start, Stride
	);
	__syncthreads();
	
	// NOTE: update weights
	MatrixAddCore(Weights, WeightsDelta, Weights, Start, Stride);
	__syncthreads();

	// NOTE: calculate bias delta
	matrix* Bias = &DenseLayer->Bias;
	matrix* BiasDelta = &TrainData->BiasDelta;
	MatrixMeanCore(NextLayerGradient, BiasDelta, Start, Stride);
	__syncthreads();
	MatrixScalarMultCore(
		TrainData->LearningRate, BiasDelta, BiasDelta, Start, Stride
	);
	__syncthreads();
	
	// NOTE: update bias
	MatrixAddCore(Bias, BiasDelta, Bias, Start, Stride);
}

__global__
void CudaDenseBackThread(
	matrix* Inputs,
	matrix* NextLayerGradient,
	dense_layer* DenseLayer,
	dense_layer_train_data* TrainData
)
{
	uint32_t Start = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t Stride = gridDim.x * blockDim.x;

	CudaDenseBackCore(
		Inputs, NextLayerGradient, DenseLayer, TrainData, Start, Stride
	);
}

void CudaDenseBack(
	matrix* Inputs,
	matrix* NextLayerGradient,
	dense_layer* DenseLayer,
	dense_layer_train_data* TrainData
)
{
	int Device = 0;
	uint32_t BlockSize = GetBlockSize(Device);
	uint32_t NumBlocks = GetNumBlocks(
		GetMatrixArrayCount(&DenseLayer->Weights), BlockSize, Device
	);
	CudaDenseBackThread<<<NumBlocks, BlockSize>>>(
		Inputs,
		NextLayerGradient,
		DenseLayer,
		TrainData
	);
	hipDeviceSynchronize();
}

void CudaAllocReluTrain(
	relu_train_data** Result, uint32_t BatchSize, uint32_t InputDim
)
{
	hipMallocManaged(Result, sizeof(relu_train_data));
	relu_train_data* TrainData = *Result;
	*TrainData = {};
	CudaInitMatrix(&TrainData->LayerGradient, BatchSize, InputDim);
}

void CudaFreeReluTrain(relu_train_data* TrainData)
{
	CudaFreeMatrixData(&TrainData->LayerGradient);
	free(TrainData);
}

__global__
void CudaReluForwardThread(matrix* Inputs, matrix* Outputs)
{
	uint32_t Start = blockIdx.x * blockDim.x + threadIdx.x;  
	uint32_t Stride = blockDim.x * gridDim.x;

	ReluForwardCore(Inputs, Outputs, Start, Stride);
}

void CudaReluForward(matrix* Inputs, matrix* Outputs)
{
	assert(Inputs->NumRows == Outputs->NumRows);
	assert(Inputs->NumColumns == Outputs->NumColumns);

	int Device = 0;
	uint32_t BlockSize = GetBlockSize(Device);
	uint32_t NumBlocks = GetNumBlocks(
		GetMatrixArrayCount(Inputs), BlockSize, Device
	);
	CudaReluForwardThread<<<NumBlocks, BlockSize>>>(Inputs, Outputs);
	hipDeviceSynchronize();
}

__global__
void CudaReluBackThread(
	matrix* Inputs, matrix* NextLayerGradient, matrix* LayerGradient
)
{
	uint32_t Start = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t Stride = gridDim.x * blockDim.x;
	ReluBackCore(
		Inputs,
		NextLayerGradient,
		LayerGradient,
		Start,
		Stride
	);
}

void CudaReluBack(
	matrix* Inputs, matrix* NextLayerGradient, relu_train_data* TrainData
)
{
	int Device = 0;
	uint32_t BlockSize = GetBlockSize(Device);
	uint32_t NumBlocks = GetNumBlocks(
		GetMatrixArrayCount(Inputs), BlockSize, Device
	);
	CudaReluBackThread<<<NumBlocks, BlockSize>>>(
		Inputs, NextLayerGradient, &TrainData->LayerGradient
	);
	hipDeviceSynchronize();
}

__device__
float CudaMseForwardCore(
	matrix* Predictions,
	matrix* Labels,
	float* Results,
	uint32_t ResultsCount,
	uint32_t Start,
	uint32_t Stride
)
{
	float ThreadResult = 0.0f;
	if(Start < ResultsCount)
	{
		ThreadResult = MseForwardCore(Predictions, Labels, Start, ResultsCount);
	}
	__syncthreads();
	
	if(Start >= ResultsCount)
	{
		// NOTE: no guarantee we'll have <= threads than results entries
		// NOTE: can't return earlier b/c of the syncthreads call
		return 0.0f;
	}
	else
	{
		Results[Start] = ThreadResult;
	}

	// NOTE: single-threaded summation
	// TODO: could try a divide-and conquer algorithm for fast summation
	float Result = 0.0f;	
	if(Start == 0)
	{
		for(int Index = 0; Index < ResultsCount; Index++)
		{
			Result += Results[Index];
		}

		Result /= (2.0f * Predictions->NumRows);
	}
	return Result;
}

__global__
void CudaMseForwardThread(
	matrix* Predictions,
	matrix* Labels,
	float* GlobalResult,
	uint32_t ResultsCount
)
{
	extern __shared__ float Results[];

	uint32_t Start = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t Stride = gridDim.x * blockDim.x;

	float Result = CudaMseForwardCore(
		Predictions, Labels, Results, ResultsCount, Start, Stride
	);
	if(Start == 0)
	{
		*GlobalResult = Result;
	}
}

float CudaMseForward(matrix* Predictions, matrix* Labels)
{
	int Device = 0;
	uint32_t BlockSize = GetBlockSize(Device);
	uint32_t NumBlocks = GetNumBlocks(
		GetMatrixArrayCount(Predictions), BlockSize, Device
	);
	float* Mse;
	hipMallocManaged(&Mse, sizeof(float));
	uint32_t ResultsCount = NumBlocks * BlockSize;
	size_t MemorySize = sizeof(float) * ResultsCount;
	CudaMseForwardThread<<<NumBlocks, BlockSize, MemorySize>>>(
		Predictions, Labels, Mse, ResultsCount
	);
	hipDeviceSynchronize();
	float Result = *Mse;
	hipFree(Mse);
	return Result;
}

__device__
void CudaMseBackCore(
	matrix* Predictions,
	matrix* Labels,
	mse_train_data* TrainData,
	uint32_t Start,
	uint32_t Stride
)
{
	MatrixSubtractCore(
		Labels, Predictions, &TrainData->LayerGradient, Start, Stride
	);
	__syncthreads();
	MatrixScalarMultCore(
		1.0f / Predictions->NumColumns,
		&TrainData->LayerGradient,
		&TrainData->LayerGradient,
		Start,
		Stride
	);
}

__global__
void CudaMseBackThread(
	matrix* Predictions,
	matrix* Labels,
	mse_train_data* TrainData
)
{
	uint32_t Start = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t Stride = gridDim.x * blockDim.x;

	CudaMseBackCore(Predictions, Labels, TrainData, Start, Stride);
}

void CudaMseBack(
	matrix* Predictions,
	matrix* Labels,
	mse_train_data* TrainData
)
{
	int Device = 0;
	uint32_t BlockSize = GetBlockSize(Device);
	uint32_t NumBlocks = GetNumBlocks(
		GetMatrixArrayCount(&TrainData->LayerGradient), BlockSize, Device
	);
	CudaMseBackThread<<<NumBlocks, BlockSize>>>(
		Predictions, Labels, TrainData
	);
	hipDeviceSynchronize();
}

void CudaAllocNeuralNet(
	neural_net** Result,
	uint32_t BatchSize,
	uint32_t InputDim,
	uint32_t CpuThreads
)
{
	hipMallocManaged(Result, sizeof(neural_net));
	neural_net* NeuralNet = *Result;
	*NeuralNet = {};
	NeuralNet->BatchSize = BatchSize;
	NeuralNet->InputDim = InputDim;
	// TODO: delete this op job allocation
	AllocMatrixOpJobs((matrix_op_jobs**) &NeuralNet->MatrixOpJobs, CpuThreads);
}

uint32_t CudaAddLayerLink(neural_net* NeuralNet, layer_type LayerType)
{
	layer_link* LayerLink = NULL;
	hipMallocManaged(&LayerLink, sizeof(layer_link));

	*LayerLink = {};
	LayerLink->Type = LayerType;
	uint32_t InputDim = NeuralNet->LastLink->Output->NumColumns;
	NeuralNet->LastLink->Next = LayerLink;
	LayerLink->Previous = NeuralNet->LastLink;
	LayerLink->Next = NULL;
	NeuralNet->LastLink = LayerLink;
	NeuralNet->NumLayers++;

	return InputDim;
}

void CudaFreeLayerLink(layer_link* LayerLink)
{
	if(LayerLink->Output != NULL)
	{
		CudaFreeMatrix(LayerLink->Output);
	}
	hipFree(LayerLink);
}

void CudaAddDense(
	neural_net* NeuralNet, uint32_t OutputDim, dense_layer* DenseLayer = NULL
)
{
	layer_link* LayerLink = NULL;
	hipMallocManaged(&LayerLink, sizeof(layer_link));

	*LayerLink = {};
	uint32_t InputDim;
	if(NeuralNet->NumLayers == 0)
	{
		InputDim = NeuralNet->InputDim;
		NeuralNet->FirstLink = LayerLink;
		NeuralNet->LastLink = LayerLink;
		LayerLink->Next = NULL;
		LayerLink->Previous = NULL;
	}
	else
	{
		InputDim = NeuralNet->LastLink->Output->NumColumns;
		LayerLink->Previous = NeuralNet->LastLink;
		NeuralNet->LastLink->Next = LayerLink;
		NeuralNet->LastLink = LayerLink;
	}

	LayerLink->Type = LayerType_Dense;
	if(DenseLayer)
	{
		LayerLink->Data = DenseLayer;
	}
	else
	{
		CudaAllocDenseLayer(
			(dense_layer**) &LayerLink->Data, 
			InputDim,
			OutputDim
		);
	}
	CudaAllocMatrix(&LayerLink->Output, NeuralNet->BatchSize, OutputDim);

	NeuralNet->NumLayers++;
}

void CudaAddRelu(neural_net* NeuralNet)
{
	uint32_t InputDim = CudaAddLayerLink(NeuralNet, LayerType_Relu);
	layer_link* LayerLink = NeuralNet->LastLink;

	CudaAllocMatrix(&LayerLink->Output, NeuralNet->BatchSize, InputDim);
}

void CudaAddMeanSquared(neural_net* NeuralNet)
{
	CudaAddLayerLink(NeuralNet, LayerType_Mse);
}

void CudaFreeNeuralNet(neural_net* NeuralNet)
{
	layer_link* LayerLink = NeuralNet->FirstLink;
	for(
		uint32_t LayerIndex = 0;
		LayerIndex < NeuralNet->NumLayers;
		LayerIndex++
	)
	{
		switch(LayerLink->Type)
		{
			case(LayerType_Dense):
			{
				dense_layer* DenseLayer = (dense_layer*) LayerLink->Data;
				CudaFreeDenseLayer(DenseLayer);
				break;
			}
			case(LayerType_Relu):
			{				
				break;
			}
			case(LayerType_Softmax):
			{
				// TODO: NOT IMPLEMENTED
				break;
			}
			case(LayerType_CrossEntropy):
			{
				// TODO: NOT IMPLEMENTED
				break;
			}
			case(LayerType_Mse):
			{
				break;
			}
			default:
			{				
				break;
			}
		}

		layer_link* Next = LayerLink->Next;
		CudaFreeLayerLink(LayerLink);
		LayerLink = Next;
	}
}

void CudaResizedNeuralNet(
	neural_net** Result, neural_net* Source, uint32_t NewBatchSize
)
{
	// NOTE: this is needed b/c the result from each layer is preallocated
	// CONT: so we can't use different batch sizes with the same neural net.
	// CONT: Instead of copying all the data, I am using this function to 
	// CONT: create the new output matrices and reusing the dense_layer structs
	// CONT: from the Source net. This is a valuable approach for situations 
	// CONT: where you are testing in a loop, e.g. if you check the full-batch
	// CONT: loss after doing all the mini batches in an epoch. It's also a 
	// CONT: slightly smaller memory profile

	matrix_op_jobs* MatrixOpJobs = Source->MatrixOpJobs;
	CudaAllocNeuralNet(
		Result, NewBatchSize, Source->InputDim, MatrixOpJobs->NumThreads
	);
	neural_net* NeuralNet = *Result;

	layer_link* LayerLink = Source->FirstLink;
	for(
		uint32_t LayerIndex = 0;
		LayerIndex < Source->NumLayers;
		LayerIndex++
	)
	{
		switch(LayerLink->Type)
		{
			case(LayerType_Dense):
			{
				dense_layer* DenseLayer = (dense_layer*) LayerLink->Data;
				CudaAddDense(
					NeuralNet, DenseLayer->Weights.NumColumns, DenseLayer
				);
				break;
			}
			case(LayerType_Relu):
			{
				CudaAddRelu(NeuralNet);
				break;
			}
			case(LayerType_Softmax):
			{
				// TODO: NOT IMPLEMENTED
				assert(false);
				break;
			}
			case(LayerType_CrossEntropy):
			{
				// TODO: NOT IMPLEMENTED
				assert(false);
				break;
			}
			case(LayerType_Mse):
			{
				CudaAddMeanSquared(NeuralNet);
				break;
			}
			default:
			{				
				break;
			}
		}
		LayerLink = LayerLink->Next;
	}
}

void CudaAllocMseTrainData(
	mse_train_data** Result, uint32_t BatchSize, uint32_t PredictionDim
)
{
	hipMallocManaged(Result, sizeof(mse_train_data));
	mse_train_data* TrainData = *Result;
	*TrainData = {};
	CudaInitMatrix(&TrainData->LayerGradient, BatchSize, PredictionDim);
}

void CudaFreeMseTrainData(mse_train_data* TrainData)
{
	CudaFreeMatrixData(&TrainData->LayerGradient);
	hipFree(TrainData);
}

void CudaFreeResizedNeuralNet(neural_net* NeuralNet)
{
	layer_link* LayerLink = NeuralNet->FirstLink;
	for(
		uint32_t LayerIndex = 0;
		LayerIndex < NeuralNet->NumLayers;
		LayerIndex++
	)
	{
		layer_link* Next = LayerLink->Next;
		CudaFreeLayerLink(LayerLink);
		LayerLink = Next;
	}
}

__device__
float CudaNeuralNetForwardCore(
	neural_net* NeuralNet,
	matrix* Inputs,
	matrix* Labels,
	float* MseResults,
	uint32_t MseResultsCount,
	uint32_t ThreadIndex,
	uint32_t ThreadCount
)
{
	matrix* Outputs = NULL;
	layer_link* LayerLink = NeuralNet->FirstLink;
	float Loss = -1.0f;
	uint32_t NumLayers = NeuralNet->NumLayers;
	for(
		uint32_t LayerIndex = 0;
		LayerIndex < NumLayers;
		LayerIndex++
	)
	{
		Outputs = LayerLink->Output;
		
		switch(LayerLink->Type)
		{
			case(LayerType_Dense):
			{
				CudaDenseForwardCore(
					Inputs,
					(dense_layer*) LayerLink->Data,
					Outputs,
					ThreadIndex,
					ThreadCount
				);
				break;
			}
			case(LayerType_Relu):
			{
				ReluForwardCore(Inputs, Outputs, ThreadIndex, ThreadCount);
				break;
			}
			case(LayerType_Softmax):
			{
				// TODO: not implemented
				break;
			}

			// NOTE: for NNs with loss layers, predictions must be captured 
			// CONT: with inputs the end of the loop since outputs 
			// CONT: will be updated to NULL
			case(LayerType_CrossEntropy):
			{
				// TODO: not implemented
				break;
			}
			case(LayerType_Mse):
			{
				if(Labels != NULL)
				{
					Loss = CudaMseForwardCore(
						Inputs,
						Labels,
						MseResults,
						MseResultsCount,
						ThreadIndex,
						ThreadCount
					);
				}
				break;
			}

			default:
			{				
				break;
			}
		}
		Inputs = Outputs;
		LayerLink = LayerLink->Next;
		__syncthreads();
	}

	return Loss;
}

__global__
void CudaNeuralNetForwardThread(
	neural_net* NeuralNet,
	matrix* Inputs,
	matrix* Labels,
	float* LossResult,
	uint32_t MseResultsCount
)
{
	extern __shared__ float MseResults[];

	uint32_t ThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t NumThreads = gridDim.x * blockDim.x;
	float Loss = CudaNeuralNetForwardCore(
		NeuralNet,
		Inputs,
		Labels,
		MseResults,
		MseResultsCount,
		ThreadIndex,
		NumThreads
	);
	__syncthreads();
	if(ThreadIndex == 0 && LossResult)
	{
		*LossResult = Loss;
	}
}

hipError_t CudaNeuralNetForward(
	neural_net* NeuralNet,
	matrix* Inputs,
	matrix* Labels,
	matrix** Predictions,
	float* LossResult
)
{
	int Device = 0;
	uint32_t BlockSize = GetBlockSize(Device);
	uint32_t NumBlocks = GetMaxNumBlocks(Device);
	
	uint32_t MseResultsCount; 
	if(Labels)
	{
		MseResultsCount = Labels->NumRows;	
	}
	else
	{
		MseResultsCount = 1;
	}
	size_t MemorySize = MseResultsCount * sizeof(float);
	CudaNeuralNetForwardThread<<<NumBlocks, BlockSize, MemorySize>>>(
		NeuralNet, Inputs, Labels, LossResult, MseResultsCount
	);
	hipError_t Error = hipDeviceSynchronize();

	if(Predictions)
	{
		*Predictions = GetOutput(NeuralNet);
	}
	return Error;
}

void CudaAllocNeuralNetTrainer(
	neural_net_trainer** Result,
	neural_net* NeuralNet,
	float LearningRate,
	layer_type LossLayer
)
{
	switch(LossLayer)
	{
		case(LayerType_Mse):
		{
			CudaAddMeanSquared(NeuralNet);
			break;
		}
		case(LayerType_CrossEntropy):
		{
			// TODO: implement
			assert(false);
			break;
		}
		default:
		{
			break;
		}
	}

	hipMallocManaged(Result, sizeof(neural_net_trainer));
	neural_net_trainer* Trainer = *Result;
	*Trainer = {};
	Trainer->NeuralNet = NeuralNet;
	hipMallocManaged(
		&Trainer->TrainDataArray, NeuralNet->NumLayers * sizeof(void*)
	);
	void** TrainDataArray = Trainer->TrainDataArray;
	memset(TrainDataArray, 0, NeuralNet->NumLayers * sizeof(void*));
	
	layer_link* LayerLink = NeuralNet->FirstLink;
	for(
		uint32_t LayerIndex = 0;
		LayerIndex < NeuralNet->NumLayers;
		LayerIndex++
	)
	{
		switch(LayerLink->Type)
		{
			case(LayerType_Dense):
			{
				CudaAllocDenseLayerTrain(
					(dense_layer_train_data**) &TrainDataArray[LayerIndex],
					(dense_layer*) LayerLink->Data,
					LearningRate,
					NeuralNet->BatchSize
				);
				break;
			}
			case(LayerType_Relu):
			{
				layer_link* PreviousLayer = LayerLink->Previous;
				matrix* PrevOutputs = PreviousLayer->Output;
				CudaAllocReluTrain(
					(relu_train_data**) &TrainDataArray[LayerIndex],
					NeuralNet->BatchSize,
					PrevOutputs->NumColumns
				);
				break;
			}
			case(LayerType_Softmax):
			{
				// TODO: implement
				assert(false);
				break;
			}
			case(LayerType_CrossEntropy):
			{
				// TODO: implement
				assert(false);
				// layer_link* PreviousLayer = LayerLink->Previous;
				// softmax_layer* SoftmaxLayer = (softmax_layer*)(
				// 	PreviousLayer->Data
				// );

				// AllocCrossEntropySoftmaxTrain(
				// 	(
				// 		(cross_entropy_softmax_train_data**) 
				// 		&TrainDataArray[LayerIndex]
				// 	),
				// 	SoftmaxLayer
				// );
				break;
			}
			case(LayerType_Mse):
			{
				layer_link* PreviousLayer = LayerLink->Previous;
				matrix* PrevOutputs = PreviousLayer->Output;
				CudaAllocMseTrainData(
					(mse_train_data**) &TrainDataArray[LayerIndex],
					NeuralNet->BatchSize,
					PrevOutputs->NumColumns
				);
				break;
			}
			default:
			{				
				break;
			}
		}
		LayerLink = LayerLink->Next;
	}

	*Result = Trainer;
}

void CudaAllocNeuralNetTrainer(
	neural_net_trainer** Result,
	neural_net* NeuralNet,
	float LearningRate,
	layer_type LossLayer,
	uint32_t MiniBatchSize,
	uint32_t OutputDim
)
{
	// NOTE: function also allocates minibatch matrices
	CudaAllocNeuralNetTrainer(Result, NeuralNet, LearningRate, LossLayer);
	neural_net_trainer* Trainer = *Result;
	CudaAllocMatrix(&Trainer->MiniBatchData, MiniBatchSize, NeuralNet->InputDim);
	CudaAllocMatrix(&Trainer->MiniBatchLabels, MiniBatchSize, OutputDim);
}

void CudaFreeNeuralNetTrainer(neural_net_trainer* Trainer)
{
	// NOTE: trainers should be freed before their NNs
	neural_net* NeuralNet = Trainer->NeuralNet;
	void** TrainDataArray = Trainer->TrainDataArray;
	layer_link* LayerLink = NeuralNet->FirstLink;
	for(
		uint32_t LayerIndex = 0;
		LayerIndex < NeuralNet->NumLayers;
		LayerIndex++
	)
	{
		switch(LayerLink->Type)
		{
			case(LayerType_Dense):
			{
				CudaFreeDenseLayerTrain(
					(dense_layer_train_data*) TrainDataArray[LayerIndex]					
				);
				break;
			}
			case(LayerType_Relu):
			{
				CudaFreeReluTrain(
					(relu_train_data*) TrainDataArray[LayerIndex]
				);
				break;
			}
			case(LayerType_Softmax):
			{
				// TODO: implement
				break;
			}
			case(LayerType_CrossEntropy):
			{
				// TODO: implement
				break;
			}
			case(LayerType_Mse):
			{
				CudaFreeMseTrainData(
					(mse_train_data*) TrainDataArray[LayerIndex]
				);
				break;
			}
			default:
			{
				break;
			}
		}
		LayerLink = LayerLink->Next;
	}

	hipFree(TrainDataArray);
	hipFree(Trainer);
}

void CudaTrainNeuralNet(
	neural_net_trainer* Trainer,
	neural_net* NeuralNet,
	matrix* Inputs,
	matrix* Labels,
	uint32_t Epochs,
	bool ShouldInitDenseLayers = true,
	bool PrintStatus = false,
	float* Loss = NULL
)
{
	if(ShouldInitDenseLayers)
	{
		InitDenseLayers(NeuralNet);
	}

	layer_link* LayerLink;

	bool FreeAtEnd = false;
	if(Loss == NULL)
	{
		hipMallocManaged(&Loss, sizeof(float));
	}

	for(uint32_t Epoch = 0; Epoch < Epochs; Epoch++)
	{
		matrix* Predictions = NULL;
		CudaNeuralNetForward(
			NeuralNet,
			Inputs,
			Labels,
			&Predictions,
			Loss
		);
		if(PrintStatus)
		{
			printf("Epoch %d Loss: %f\n", Epoch, *Loss);
		}

		matrix* NextLayerGradient = NULL;
		LayerLink = NeuralNet->LastLink;
		for(
			int32_t LayerIndex = ((int32_t) NeuralNet->NumLayers) - 1;
			LayerIndex >= 0;
			LayerIndex--
		)
		{
			void* TrainData = Trainer->TrainDataArray[LayerIndex];
			layer_link* PreviousLayer = LayerLink->Previous;
			matrix* LayerInputs;
			if(PreviousLayer != NULL)
			{
				LayerInputs = PreviousLayer->Output;
			}
			else
			{
				LayerInputs = Inputs;
			}
			switch(LayerLink->Type)
			{
				case(LayerType_Dense):
				{
					dense_layer_train_data* DenseTrain = (
						(dense_layer_train_data*) TrainData
					);
					CudaDenseBack(
						LayerInputs,
						NextLayerGradient,
						(dense_layer*) LayerLink->Data,
						DenseTrain
					);
					NextLayerGradient = &DenseTrain->LayerGradient;
					break;
				}
				case(LayerType_Relu):
				{
					relu_train_data* ReluTrain = (relu_train_data*) TrainData;
					CudaReluBack(
						LayerInputs,
						NextLayerGradient,
						ReluTrain
					);
					NextLayerGradient = &ReluTrain->LayerGradient;
					break;
				}
				case(LayerType_Softmax):
				{
					break;
				}
				case(LayerType_Mse):
				{
					mse_train_data* MseTrain = (mse_train_data*) TrainData;

					CudaMseBack(
						Predictions,
						Labels,
						MseTrain
					);
					NextLayerGradient = &MseTrain->LayerGradient;
					break;
				}
				case(LayerType_CrossEntropy):
				{					
					break;
				}
				default:
				{
					break;
				}
			}
			LayerLink = PreviousLayer;
		}
	}

	if(FreeAtEnd)
	{
		hipFree(Loss);
	}
}

void CudaTrainNeuralNetMiniBatch(
	neural_net_trainer* Trainer,
	neural_net* NeuralNet,
	matrix* Inputs,
	matrix* Labels,
	uint32_t Epochs,
	bool ShouldInitDenseLayers = true,
	bool PrintStatus = false,
	float TrainingAccuracyThreshold = 1.1f,
	float LossThreshold = -1.0f,
	neural_net* FullBatchNnViewer = NULL,
	float* Loss = NULL
)
{
	// NOTE: Train with minibatches sampled from Inputs
	assert(Trainer->MiniBatchData != NULL);
	assert(Trainer->MiniBatchLabels != NULL);

	matrix* MiniBatchData = Trainer->MiniBatchData;
	matrix* MiniBatchLabels = Trainer->MiniBatchLabels;
	uint32_t TrainingSamples = Inputs->NumRows;
	uint32_t MiniBatchSize = MiniBatchData->NumRows;
	
	if(ShouldInitDenseLayers)
	{
		InitDenseLayers(NeuralNet);
	}

	int_shuffler IntShuffler = MakeIntShuffler(TrainingSamples);

	for(uint32_t Epoch = 0; Epoch < Epochs; Epoch++)
	{
		ShuffleInts(&IntShuffler);
		for(
			uint32_t BatchIndex = 0;
			BatchIndex < TrainingSamples / MiniBatchSize;
			BatchIndex++
		)
		{
			// NOTE: create mini batch
			uint32_t IndexHandleStart = BatchIndex * MiniBatchSize;
			for(
				uint32_t IndexHandle = IndexHandleStart;
				IndexHandle < (IndexHandleStart + MiniBatchSize);
				IndexHandle++
			)
			{
				int RowToGet = IntShuffler.Result[IndexHandle];
				float* DataRow = GetMatrixRow(Inputs, RowToGet);
				float* LabelsRow = GetMatrixRow(Labels, RowToGet);

				float* MiniBatchDataRow = GetMatrixRow(
					MiniBatchData, IndexHandle - IndexHandleStart
				);
				float* MiniBatchLabelRow = GetMatrixRow(
					MiniBatchLabels, IndexHandle - IndexHandleStart
				);

				memcpy(
					MiniBatchDataRow,
					DataRow,
					MiniBatchData->NumColumns * sizeof(float)
				);
				memcpy(
					MiniBatchLabelRow,
					LabelsRow,
					MiniBatchLabels->NumColumns * sizeof(float)
				);
			}

			// NOTE: train on mini batch
			CudaTrainNeuralNet(
				Trainer,
				NeuralNet,
				MiniBatchData,
				MiniBatchLabels,
				1,
				false,
				false,
				Loss
			);
		}

		NeuralNetForward(
			FullBatchNnViewer,
			Inputs,
			Labels,
			NULL,
			Loss
		);
		float TrainingAccuracy = TopOneAccuracy(
			FullBatchNnViewer, Inputs, Labels
		);
		if(PrintStatus)
		{
			printf(
				"Epoch %d Loss, Accuracy: %f, %f\n",
				Epoch,
				*Loss,
				TrainingAccuracy
			);
		}
		if(
			TrainingAccuracy >= TrainingAccuracyThreshold ||
			*Loss <= LossThreshold
		)
		{
			break;
		}
	}
}